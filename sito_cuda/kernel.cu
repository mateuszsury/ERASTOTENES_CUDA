#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>

__global__ void sieve(long long* numbers, long long n, int blockSize, int numChunks) {
    int chunkId = blockIdx.x; // Identyfikator bloku w wymiarze X
    int threadId = threadIdx.x; // Identyfikator watku w wymiarze X
    long long start = chunkId * blockSize + threadId + 2; // Początkowy indeks przetwarzany przez wątek w danym bloku
    long long step = numChunks * blockSize; // Odstęp między liczbami przetwarzanymi przez dwa kolejne wątki

    for (long long index = start; index <= n; index += step) {
        if (numbers[index] != 0) {
            for (long long i = 2 * index; i <= n; i += index) {
                numbers[i] = 0;
            }
        }
    }
}

int main() {
    long long n = 1000000000;  // Liczba pierwszych do obliczenia
    int blockSize = 256;  // Rozmiar bloku wątków CUDA
    int numChunks = 8;  // Liczba fragmentów do równoczesnego przetwarzania
    int numBlocks = numChunks;
    long long* h_numbers = new long long[n + 1];  // Tablica na CPU do algorytmu sita

    // Inicjalizacja tablicy na CPU
    for (long long i = 2; i <= n; i++) {
        h_numbers[i] = i;
    }

    // Inicjalizacja timerów CUDA
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Rozpoczęcie pomiaru czasu
    hipEventRecord(start);

    // Alokuje pamięć na GPU
    long long* d_numbers;
    hipMalloc((void**)&d_numbers, (n + 1) * sizeof(long long));

    // Kopiuje dane z CPU do GPU
    hipMemcpy(d_numbers, h_numbers, (n + 1) * sizeof(long long), hipMemcpyHostToDevice);

    // Uruchamia kernel CUDA z wieloma blokami
    sieve << <numBlocks, blockSize >> > (d_numbers, n, blockSize, numChunks);



    // Kopiuje wyniki z GPU na CPU
    hipMemcpy(h_numbers, d_numbers, (n + 1) * sizeof(long long), hipMemcpyDeviceToHost);

    // Zakończenie pomiaru czasu
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Oblicza sumę liczb pierwszych
    long long sum = 0;

    std::cout << "Liczby pierwsze: ";
    for (long long i = 2; i <= n; i++) {
        if (h_numbers[i] != 0) {
            sum += h_numbers[i];
            //std::cout << h_numbers[i] << " ";
        }
    }
    std::cout << std::endl;

    std::cout << "Suma " << n << " liczb pierwszych: " << sum << std::endl;

    // Obliczanie czasu trwania
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Czas obliczen na GPU: " << milliseconds << " ms" << std::endl;

    // Zwolnienie pamięci i timerów
    delete[] h_numbers;
    hipFree(d_numbers);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
